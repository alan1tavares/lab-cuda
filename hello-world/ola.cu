
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void olaMundo(void){
    printf("Ola GPU!\n");
}

int main(void) {
    printf("Ola CPU!\n");
    olaMundo <<<1,10>>>();
    hipDeviceReset();
    return 0;
}